#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>

#include "FortranArray2D.cu"

// Working precision
typedef double wp_t;

// <<<number_of_blocks, size_of_block>>>

struct GridConstants {
    FortranArray2D<wp_t, 1, 1> * e1t = nullptr;
    FortranArray2D<wp_t, 1, 1> * e2t = nullptr;
    FortranArray2D<wp_t, 0, 1> * e1u = nullptr;
    FortranArray2D<wp_t, 0, 1> * e2u = nullptr;

    FortranArray2D<wp_t, 0, 0> * e1f = nullptr;
    FortranArray2D<wp_t, 0, 0> * e2f = nullptr;
    FortranArray2D<wp_t, 1, 0> * e1v = nullptr;
    FortranArray2D<wp_t, 1, 0> * e2v = nullptr;

    FortranArray2D<wp_t, 1, 1> * e12t = nullptr;
    FortranArray2D<wp_t, 0, 1> * e12u = nullptr;
    FortranArray2D<wp_t, 1, 0> * e12v = nullptr;

    FortranArray2D<wp_t, 0, 1> * gphiu = nullptr;
    FortranArray2D<wp_t, 1, 0> * gphiv = nullptr;
    FortranArray2D<wp_t, 0, 0> * gphif = nullptr;

    FortranArray2D<wp_t, 1, 1> * xt = nullptr;
    FortranArray2D<wp_t, 1, 1> * yt = nullptr;

    FortranArray2D<wp_t, 1, 1> * ht = nullptr;
    FortranArray2D<wp_t, 0, 1> * hu = nullptr;
    FortranArray2D<wp_t, 0, 1> * hv = nullptr;

    // -1 = Water cell outside computational domain
    //  0 = Land cell
    //  1 = Water cell inside computational domain
    FortranArray2D<int, 0, 0> * pt = nullptr;
};

struct SimulationVariables {
    // Sea surface height - current values.
    FortranArray2D<wp_t, 1, 1> * sshn = nullptr;
    FortranArray2D<wp_t, 0, 1> * sshn_u = nullptr;
    FortranArray2D<wp_t, 1, 0> * sshn_v = nullptr;

    // Sea surface height - next step's values
    FortranArray2D<wp_t, 1, 1> * ssha = nullptr;
    FortranArray2D<wp_t, 0, 1> * ssha_u = nullptr;
    FortranArray2D<wp_t, 1, 0> * ssha_v = nullptr;

    // Velocities - current values
    FortranArray2D<wp_t, 0, 1> * un = nullptr;
    FortranArray2D<wp_t, 1, 0> * vn = nullptr;

    // Velocities - next step's values
    FortranArray2D<wp_t, 0, 1> * ua = nullptr;
    FortranArray2D<wp_t, 1, 0> * va = nullptr;
};

struct ModelParameters {
    // Number of columns in modle grid
    int jpi = 0;

    // Number of rows in model grid
    int jpj = 0;

    // Grid size in x and y directions (m)
    wp_t dx = 0;
    wp_t dy = 0;

    // Constant depth (m)
    wp_t dep_const = 0.0;

    // First time step
    int nit000 = 0;

    // Final time step
    int nitend = 0;

    // Interval on which to save results
    int irecord = 0;

    // Size of time step (s)
    wp_t rdt = 0.0;

    // Bottom friction coefficient
    wp_t cbfr = 0.0;

    // Horizontal kinematic viscosity coefficient
    wp_t visc = 0.0;
};

__global__ void k_setup_model_params(
    const int jpi,
    const int jpj,
    const wp_t dx,
    const wp_t dy,
    const wp_t dep_const,
    const int nit000,
    const int nitend,
    const int irecord,
    const wp_t rdt,
    const wp_t cbfr,
    const wp_t visc
);
__global__ void k_continuity();
__global__ void k_boundary_conditions();
__global__ void k_momentum();
__global__ void k_next();

void finalise();

extern "C" {
    void cuda_setup_model_params_(
        int jpi,
        int jpj,
        wp_t dx,
        wp_t dy,
        wp_t dep_const,
        int nit000,
        int nitend,
        int irecord,
        wp_t rdt,
        wp_t cbfr,
        wp_t visc
    ) {
        k_setup_model_params<<<1, 1>>>(
            jpi,
            jpj,
            dx,
            dy,
            dep_const,
            nit000,
            nitend,
            irecord,
            rdt,
            cbfr,
            visc
        );
    }

    void cuda_initialise_grid_();

    void cuda_continuity_() {
        k_continuity<<<1, 10>>>();
    }

    void cuda_boundary_conditions_() {
        k_boundary_conditions<<<1, 10>>>();
    }

    void cuda_momentum_() {
        k_momentum<<<1, 10>>>();
        hipDeviceSynchronize();
    }

    void cuda_next_() {
        k_next<<<1, 10>>>();
    }

    void cuda_finalise_() {
        finalise();
    }
};

__device__ GridConstants grid_constants;
__device__ SimulationVariables simulation_vars;
__device__ ModelParameters model_params;

void cuda_initialise_grid_() {
    const int jpi = model_params.jpi;
    const int jpj = model_params.jpj;

    if (jpi == 0 || jpj == 0) {
        fprintf(
            stderr,
            "Invalid grid size: (%d, %d); have you setup model params?",
            model_params.jpi,
            model_params.jpj
        );
    }

    // Create and allocate the grid constants 
    grid_constants.e1t = new FortranArray2D<wp_t, 1, 1>(jpi, jpj);
    grid_constants.e2t = new FortranArray2D<wp_t, 1, 1>(jpi, jpj);
    grid_constants.e1u = new FortranArray2D<wp_t, 0, 1>(jpi, jpj);
    grid_constants.e2u = new FortranArray2D<wp_t, 0, 1>(jpi, jpj);

    grid_constants.e1f = new FortranArray2D<wp_t, 0, 0>(jpi, jpj);
    grid_constants.e2f = new FortranArray2D<wp_t, 0, 0>(jpi, jpj);
    grid_constants.e1v = new FortranArray2D<wp_t, 1, 0>(jpi, jpj);
    grid_constants.e2v = new FortranArray2D<wp_t, 1, 0>(jpi, jpj);

    grid_constants.e12t = new FortranArray2D<wp_t, 1, 1> (jpi, jpj);
    grid_constants.e12u = new FortranArray2D<wp_t, 0, 1> (jpi, jpj);
    grid_constants.e12v = new FortranArray2D<wp_t, 1, 0> (jpi, jpj);

    grid_constants.gphiu = new FortranArray2D<wp_t, 0, 1>(jpi, jpj);
    grid_constants.gphiv = new FortranArray2D<wp_t, 1, 0>(jpi, jpj);
    grid_constants.gphif = new FortranArray2D<wp_t, 0, 0>(jpi, jpj);

    grid_constants.xt = new FortranArray2D<wp_t, 1, 1>(jpi, jpj);
    grid_constants.yt = new FortranArray2D<wp_t, 1, 1>(jpi, jpj);

    grid_constants.ht = new FortranArray2D<wp_t, 1, 1>(jpi, jpj);
    grid_constants.hu = new FortranArray2D<wp_t, 0, 1>(jpi, jpj);
    grid_constants.hv = new FortranArray2D<wp_t, 0, 1>(jpi, jpj);

    grid_constants.pt = new FortranArray2D<int, 0, 0>(jpi + 1, jpj + 1);

    // Create and allocate the simulation variables
    // TODO:

    const wp_t initial_value = 0.0;

    // Initialise grid constants to initial values values.
    grid_constants.ht->set_value(initial_value);
    grid_constants.hu->set_value(initial_value);
    grid_constants.hv->set_value(initial_value);

    grid_constants.xt->set_value(initial_value);
    grid_constants.yt->set_value(initial_value);

    grid_constants.pt->set_value(initial_value);

    // Initialise simulation variables to initial values.
    simulation_vars.sshn->set_value(initial_value);
    simulation_vars.sshn_u->set_value(initial_value);
    simulation_vars.sshn_v->set_value(initial_value);

    simulation_vars.ssha->set_value(initial_value);
    simulation_vars.ssha_u->set_value(initial_value);
    simulation_vars.ssha_v->set_value(initial_value);

    simulation_vars.un->set_value(initial_value);
    simulation_vars.vn->set_value(initial_value);
    simulation_vars.ua->set_value(initial_value);
    simulation_vars.va->set_value(initial_value);

    // TODO: These set the boundary conditions, we need to set the initial conditions.
}

__global__ void k_setup_model_params(
    const int jpi,
    const int jpj,
    const wp_t dx,
    const wp_t dy,
    const wp_t dep_const,
    const int nit000,
    const int nitend,
    const int irecord,
    const wp_t rdt,
    const wp_t cbfr,
    const wp_t visc
) {
    printf("Initialising model parameters on device.\n");
    model_params = {
        .jpi = jpi,
        .jpj = jpj,
        .dx = dx,
        .dy = dy,
        .dep_const = dep_const,
        .nit000 = nit000,
        .nitend = nitend,
        .irecord = irecord,
        .rdt = rdt,
        .cbfr = cbfr,
        .visc = visc,
    };
}

__global__ void k_continuity() {
    // TODO:
}

__global__ void k_momentum() {
    // TODO:
}

__global__ void k_boundary_conditions() {
    // TODO:
}

__global__ void k_next() {
    // TODO:
}

void finalise() {
    hipError_t cudaStatus;

    // TODO: hipFree all the device memory.

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA device reset failed.");
    }

    delete grid_constants.e1t;
    delete grid_constants.e2t;
    delete grid_constants.e1u;
    delete grid_constants.e2u;

    delete grid_constants.e1f;
    delete grid_constants.e2f;
    delete grid_constants.e1v;
    delete grid_constants.e2v;

    delete grid_constants.e12t;
    delete grid_constants.e12u;
    delete grid_constants.e12v;

    delete grid_constants.gphiu;
    delete grid_constants.gphiv;
    delete grid_constants.gphif;

    delete grid_constants.xt;
    delete grid_constants.yt;

    delete grid_constants.ht;
    delete grid_constants.hu;
    delete grid_constants.hv;

    delete grid_constants.pt;

    // TODO: Delete simulation params arrays.
}
